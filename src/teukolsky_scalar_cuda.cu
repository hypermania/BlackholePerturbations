#include "teukolsky_scalar_cuda.cuh"
#include "cuda_wrapper.cuh"
#include "pde_cuda_kernel.cuh"

CudaTeukolskyScalarPDE::CudaTeukolskyScalarPDE(Param param_) : param(param_) {
  using namespace Eigen;
  //using namespace Teukolsky;
    
  const Scalar rast_min = param.rast_min;
  const Scalar rast_max = param.rast_max;
  const auto N = param.N;
  const Scalar M = param.M;
  const Scalar a = param.a;
  const auto s = param.s;
  const auto l_max = param.l_max;

  grid_size = N + 1;
  lm_size = (l_max + 1) * (l_max + 1);
    
  const Scalar h = (rast_max - rast_min) / (N - 1);

  // Load coupling mapping info
  psi_lm_map = Teukolsky::make_coupling_info_map(Teukolsky::psi_lm_coupling_info_scalar, l_max);
  dr_psi_lm_map = Teukolsky::make_coupling_info_map(Teukolsky::dr_psi_lm_coupling_info_scalar, l_max);
  drdr_psi_lm_map = Teukolsky::make_coupling_info_map(Teukolsky::drdr_psi_lm_coupling_info_scalar, l_max);
  dt_psi_lm_map = Teukolsky::make_coupling_info_map(Teukolsky::dt_psi_lm_coupling_info_scalar, l_max);

  // Compute the radial coordinate and coupling coefficients
  auto a_hp = static_cast<HighPrecisionScalar>(a);
  auto M_hp = static_cast<HighPrecisionScalar>(M);
  auto rast_min_hp = static_cast<HighPrecisionScalar>(rast_min);
  auto rast_max_hp = static_cast<HighPrecisionScalar>(rast_max);

  auto t1 = std::chrono::system_clock::now();

  std::cout << "point 0" << std::endl;
  
  auto r_hp = Teukolsky::compute_hp_r_vector(rast_min_hp, rast_max_hp, N, M_hp, a_hp);

  std::cout << "point 1" << std::endl;
  auto t2 = std::chrono::system_clock::now();

  auto coeffs_eigen = Teukolsky::compute_coeffs_scalar(a_hp, M_hp, r_hp);
  std::cout << "point 2" << std::endl;
  coeffs.resize(coeffs_eigen.size());
  std::cout << "point 3" << std::endl;
  for(size_t i = 0; i < coeffs_eigen.size(); ++i){
    coeffs[i].resize(coeffs_eigen[i].size());
    copy_vector(coeffs[i], coeffs_eigen[i]);
  }
  std::cout << "point 4" << std::endl;
  return;
  
  auto t3 = std::chrono::system_clock::now();
  std::chrono::duration<double> time_diff_1 = t2 - t1;
  std::chrono::duration<double> time_diff_2 = t3 - t2;
  std::cout << std::setw(9) << "time spent 1 = " << time_diff_1.count() << " s" << '\n';
  std::cout << std::setw(9) << "time spent 2 = " << time_diff_2.count() << " s" << '\n';


  
  //Q = [N](const Scalar t)->Vector{ return Vector::Zero(N+1); };

  // Prepare buffer for first and second derivatives of psi_lm
  drdr_psi_lm.resize(lm_size * grid_size);
  dr_psi_lm.resize(lm_size * grid_size);


  // Placeholder states
  State x(2 * lm_size * grid_size);
  State dxdt(2 * lm_size * grid_size);
  hipGraph_t system_graph = prepare_cuda_graph(x, dxdt);
  hipError_t err = hipGraphInstantiate(&system_graph_exec, system_graph, 0);
  std::cout << "(hipGraphInstantiate) err = " << err << std::endl;
  err = hipGraphDestroy(system_graph);
  std::cout << "(hipGraphDestroy) err = " << err << std::endl;
}


hipGraph_t CudaTeukolskyScalarPDE::prepare_cuda_graph(const State &x, State &dxdt)
{
  const Scalar rast_min = param.rast_min;
  const Scalar rast_max = param.rast_max;
  const auto N = param.N;
  const Scalar h = (rast_max - rast_min) / (N - 1);
  
  // Prepare CUDA graph for operator()
  hipGraph_t graph;
  hipError_t err = hipGraphCreate(&graph, 0);

  std::cout << "(GraphCreate) err = " << err << std::endl;

  hipGraphNode_t copy_time_derivative_node;
  std::vector<hipGraphNode_t> compute_derivative_nodes(2 * lm_size);
  hipGraphNode_t empty_barrier_node;
  std::vector<hipGraphNode_t> compute_dtdt_nodes(lm_size);
  
  // Add computational of derivatives into the graph

  // Copy first order time derivative
  const long long int dt_grid_begin = lm_size * grid_size;
  err = hipGraphAddMemcpyNode1D(&copy_time_derivative_node, graph, NULL, 0,
				 (void *)thrust::raw_pointer_cast(dxdt.data()),
				 (const void *)(thrust::raw_pointer_cast(x.data()) + dt_grid_begin),
				 dt_grid_begin * sizeof(thrust::complex<double>),
				 hipMemcpyDeviceToDevice);

  std::cout << "(hipGraphAddMemcpyNode1D) err = " << err << std::endl;
  
  // Compute first and second order spatial derivative  
  for(size_t lm = 0; lm < lm_size; ++lm){
    {
      auto arg1 = thrust::raw_pointer_cast(drdr_psi_lm.data() + lm * grid_size);
      auto arg2 = thrust::raw_pointer_cast(x.data() + lm * grid_size);
      int arg3 = grid_size;
      double arg4 = 1 / (h * h);

      void *ptrs[4] = {(void *)&arg1, (void *)&arg2, (void *)&arg3, (void *)&arg4};
      void **ptrs_casted = (void **)ptrs;
    
      const int threadsPerBlock = 512;
      const int numBlocks = (grid_size + threadsPerBlock - 1) / threadsPerBlock;
    
      hipKernelNodeParams node_params;
      node_params.func = (void *)CUDAKernel::drdr_complex_double_kernel;
      node_params.gridDim = dim3(numBlocks);
      node_params.blockDim = dim3(threadsPerBlock);
      node_params.sharedMemBytes = 0;
      node_params.kernelParams = ptrs_casted;
      node_params.extra = NULL;
    
      err = hipGraphAddKernelNode(&compute_derivative_nodes[lm], graph, NULL, 0, &node_params);
      std::cout << "(hipGraphAddKernelNode) err = " << err << std::endl;
    }
    
    {
      auto arg1 = thrust::raw_pointer_cast(dr_psi_lm.data() + lm * grid_size);
      auto arg2 = thrust::raw_pointer_cast(x.data() + lm * grid_size);
      int arg3 = grid_size;
      double arg4 = 1 / h;

      void *ptrs[4] = {(void *)&arg1, (void *)&arg2, (void *)&arg3, (void *)&arg4};
      void **ptrs_casted = (void **)ptrs;
    
      const int threadsPerBlock = 512;
      const int numBlocks = (grid_size + threadsPerBlock - 1) / threadsPerBlock;
    
      hipKernelNodeParams node_params;
      node_params.func = (void *)CUDAKernel::dr_complex_double_kernel;
      node_params.gridDim = dim3(numBlocks);
      node_params.blockDim = dim3(threadsPerBlock);
      node_params.sharedMemBytes = 0;
      node_params.kernelParams = ptrs_casted;
      node_params.extra = NULL;
    
      err = hipGraphAddKernelNode(&compute_derivative_nodes[lm_size + lm], graph, NULL, 0, &node_params);
      std::cout << "(hipGraphAddKernelNode) err = " << err << std::endl;
    }
  }

  // Barrier node
  hipGraphAddEmptyNode(&empty_barrier_node, graph, compute_derivative_nodes.data(), 2 * lm_size);
  std::cout << "(hipGraphAddEmptyNode) err = " << err << std::endl;

  // Assign second order time derivatives  
  for(size_t lm = 0; lm < lm_size; ++lm){
    std::vector<void *> args;
      
    for(auto [lm1, idx1] : psi_lm_map[lm]){
      thrust::complex<double> *coeff_ptr = thrust::raw_pointer_cast(coeffs[idx1].data());
      const thrust::complex<double> *var_ptr = thrust::raw_pointer_cast(x.data() + lm1 * grid_size);
      args.push_back(reinterpret_cast<void *>(coeff_ptr));
      args.push_back(const_cast<void *>(reinterpret_cast<const void *>(var_ptr)));
    }
    for(auto [lm1, idx1] : dt_psi_lm_map[lm]){
      thrust::complex<double> *coeff_ptr = thrust::raw_pointer_cast(coeffs[idx1].data());
      const thrust::complex<double> *var_ptr = thrust::raw_pointer_cast(x.data() + (lm_size + lm1) * grid_size);
      args.push_back(reinterpret_cast<void *>(coeff_ptr));
      args.push_back(const_cast<void *>(reinterpret_cast<const void *>(var_ptr)));
    }
    for(auto [lm1, idx1] : dr_psi_lm_map[lm]){
      thrust::complex<double> *coeff_ptr = thrust::raw_pointer_cast(coeffs[idx1].data());
      thrust::complex<double> *var_ptr = thrust::raw_pointer_cast(dr_psi_lm.data() + lm1 * grid_size);
      args.push_back(reinterpret_cast<void *>(coeff_ptr));
      args.push_back(reinterpret_cast<void *>(var_ptr));
    }
    for(auto [lm1, idx1] : drdr_psi_lm_map[lm]){
      thrust::complex<double> *coeff_ptr = thrust::raw_pointer_cast(coeffs[idx1].data());
      thrust::complex<double> *var_ptr = thrust::raw_pointer_cast(drdr_psi_lm.data() + lm1 * grid_size);
      args.push_back(reinterpret_cast<void *>(coeff_ptr));
      args.push_back(reinterpret_cast<void *>(var_ptr));
    }

    auto arg_lhs = thrust::raw_pointer_cast(dxdt.data() + (lm_size + lm) * grid_size);
    int grid_size_store = grid_size;
    // args.push_back(reinterpret_cast<void *>(&arg_lhs));
    // args.push_back(reinterpret_cast<void *>(&grid_size_store));
    
    // The number of terms on the RHS
    // Since the total number of arguments of the kernel is 2n, num_terms is give below
    const size_t num_terms = args.size() / 2;
    
    std::vector<void *> ptrs(args.size() + 2);
    ptrs[0] = (void *)&arg_lhs;
    ptrs[ptrs.size()-1] = (void *)&grid_size_store;
    for(size_t i = 0; i < args.size(); ++i){
      ptrs[i+1] = (void *)&args[i];
    }

    const int threadsPerBlock = 512;
    const int numBlocks = (grid_size + threadsPerBlock - 1) / threadsPerBlock;

    hipKernelNodeParams node_params;
    node_params.func = (void *)CUDAKernel::assign_lhs_2terms_complex_double_kernels[num_terms];
    node_params.gridDim = dim3(numBlocks);
    node_params.blockDim = dim3(threadsPerBlock);
    node_params.sharedMemBytes = 0;
    node_params.kernelParams = reinterpret_cast<void **>(ptrs.data());
    node_params.extra = NULL;
    
    // dtdt computation should depend on the empty barrier node
    err = hipGraphAddKernelNode(&compute_dtdt_nodes[lm], graph, &empty_barrier_node, 1, &node_params);
    std::cout << "(hipGraphAddKernelNode) err = " << err << std::endl;
  }

  size_t numNodes;
  err = hipGraphGetNodes(graph, NULL, &numNodes);
  std::cout << "(graph prepared)" << std::endl;
  std::cout << "numNodes = " << numNodes << std::endl;
  std::cout << "err = " << err << std::endl;

  return graph;
}

void CudaTeukolskyScalarPDE::operator()(const State &x, State &dxdt, const Scalar t)
{
  // const auto N = param.N;
  // const long long int dt_grid_begin = lm_size * grid_size;

  // hipMemcpy((void *)thrust::raw_pointer_cast(dxdt.data()),
  // 	     (const void *)(thrust::raw_pointer_cast(x.data()) + dt_grid_begin),
  // 	     dt_grid_begin * sizeof(thrust::complex<double>),
  // 	     hipMemcpyDeviceToDevice);

  // hipGraphExecUpdate(hipGraphExec_t hGraphExec, hipGraph_t hGraph, cudaGraphExecUpdateResultInfo* resultInfo )
  
  cudaGraphExecUpdateResultInfo update_info;
  hipGraph_t new_graph = prepare_cuda_graph(x, dxdt);
  hipGraphExecUpdate(system_graph_exec, new_graph, &update_info);
  hipGraphLaunch(system_graph_exec, 0);
  hipGraphDestroy(new_graph);
}
